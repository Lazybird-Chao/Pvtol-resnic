/**



*/



#include "../util/cudaUtil.h"
#define CUDA
#include "cudaKernel.h"
#undef CUDA


/**
 * hipLaunchKernel function
 * 
 * \brief  This function uses the specified parameters to execute the
 *         coprocessor function specified by kernel. Any parameters to
 *         the kernel should be included in params.
 * 
 * \param  kernel A string naming the kernel to execute
 * \param  dims The dimensions of the data to execute the kernel on
 * \param  nParams The number of parameters to be passed to the kernel
 * \param  params Array of parameters to be passed to the kernel
 * \param  gDim The grid dimensions of the kernel
 * \param  bDim The block dimensions of the kernel
 * \param  locMem The amount of local or shared memory to allocate for kernel
 *         execution
 * \param  stream The stream index to associate this kernel's execution and
 *         data with
 * \return None
 *
 */
extern "C" void hipLaunchKernel(reinterpret_cast<const void*>(	const char * krn), int * dims, int nParams, int * paramSizes, void ** params,
					int * gDim, int * bDim, int locMem, HTaskInfo * info) {
  hTaskLoc loc = info->location;
  int dev = info->device;
  int proc = info->process;
  size_t offset = 0;
  dim3 gridDims = dim3(gDim[HLENGTH],gDim[HWIDTH],gDim[HDEPTH]);
  dim3 blockDims = dim3(bDim[HLENGTH],bDim[HWIDTH],bDim[HDEPTH]);


  // Set the device and configure the kernel call to a stream
  cudaCheckErr(hipSetDevice(dev),__LINE__,__FILE__);
  cudaCheckErr(hipConfigureCall(gridDims, blockDims, locMem, (hipStream_t)proc),__LINE__,__FILE__);
	
  // For each kernel parameter passed in, push the kernel argument onto the stack
  //   turn into internal function
  float *s;
  hipMalloc(&s,sizeof(float)*2);
  params[0]=&s;
  for (int i = 0; i < nParams; i++) {
    cudaCheckErr(hipSetupArgument(params[i], paramSizes[i], offset),__LINE__,__FILE__);
    offset = offset + paramSizes[i];
  }
  /*
  if (!strcmp(krn,"scale")) {
    printf("Launching %s on dev:%d\n", krn, dev);
    cudaCheckErr(hipLaunchByPtr(scale),__LINE__,__FILE__);
  }*/

  cudaCheckErr(hipGetLastError(),__LINE__,__FILE__);

  if ((proc != 0) && (proc != -1)) { hipStreamSynchronize((hipStream_t)proc); }
  //hipDeviceSynchronize();
  cudaCheckErr(hipDeviceSynchronize(),__LINE__,__FILE__);
  printf("proc= %d  run to here\n", proc);
  cudaCheckErr(hipGetLastError(),__LINE__,__FILE__);
  return;
}


