/**
 *
 *  \author  $LastChangedBy$
 *  \date    $LastChangedDate$
 *  \version $LastChangedRevision$
 *  \brief   
 *
 *  $Id$
 *
 *  Author: James Brock
 */

#include "cudaUtil.h"
#define CUDA
#include "cudaKernel.h"
#undef CUDA

// Global variables
hipDeviceProp_t ** devProp;  // CUDA Device propertie structures
int nDev;                   // Number of CUDA devices available
int cudaSysInit = 0;        // System initialization flag
int sysFlags;               // System flags

/**
 * cudaCheckErr function
 * 
 * This function checks to see if any errors have occured in using CUDA, and
 * prints out the relevent error information. This function only checks the last error to occur.
 *
 * \param err The error code to check
 * \param line The line at which the error code was generated 
 * \param file The file in which the error code was generated
 * \return None
 *
 */
extern "C" void cudaCheckErr(hipError_t err, int line, char * file) {
  if (err != hipSuccess) {
    const char * err_str = hipGetErrorString(err);
    printf("CUDA ERROR %d: %s on line %d in file %s\n", err, err_str, line, file);
#ifdef KILLONERR
    exit(err);
#endif
  }
  return;
}

/**
 * initCUDA function
 * 
 * \brief This function will initialize the CUDA system, finding any devices
 *        available, and initializing any 3rd party libraries requested
 * \return None
 *
 */
extern "C" void initCUDA(int flags) {
  sysFlags = flags;
  if (cudaSysInit == 0) {
    cudaCheckErr(hipGetDeviceCount(&nDev),__LINE__,__FILE__);
    devProp = (hipDeviceProp_t**)malloc(sizeof(hipDeviceProp_t*)*nDev);
    for (int idx = 0; idx < nDev; idx++) {
      devProp[idx] = new hipDeviceProp_t;
    }
    if (sysFlags & CUBLAS_FLAG) { /* Init CUBLAS interface */ }
    if (sysFlags & CUFFT_FLAG) { /* Init CUFFT interface */ }
    if (sysFlags & CURAND_FLAG) { /* Init CURAND interface */ }
  }
  return;
}

/**
 * closeCUDA function
 * 
 * \brief  This function will clean up, releasing and deleting all of the CUDA objects
 *         that were allocated or initialized during program execution
 * \return None
 *
 */
extern "C" void closeCUDA() {
  if (sysFlags & CUFFT_FLAG) { /* Close CUFFT interface */ }
  if (sysFlags & CUBLAS_FLAG) { /* Close CUBLAS interface */ }
  if (sysFlags & CURAND_FLAG) { /* Close CURAND interface */ }
  // Delete and close main CUDA structures last
  for (int idx = 0; idx < nDev; idx++) { delete devProp[idx]; }
  free(devProp); nDev = 0;
  return;
}

/**
 * cudaInitDevice function
 * 
 * \brief  This function selects the coprocessor device and mapping and performs
 *         any initialization tasks that need to occur.
 * \param  info Heterogeneous task information structure of the device to be initialized
 * \return None
 *
 */
extern "C" void cudaInitDevice(HTaskInfo * info) {
  int idx = 0;
  hipError_t err = hipSuccess;
  int dev = info->device;
  int proc = info->process;
  hTaskLoc loc = info->location;

  // Device Management and initialization
  if (dev != -1) {
    err = hipSetDevice(dev);
    if (err != hipSuccess) {
      printf("CUDAUTIL WARNING: Could not assign to previously initialized device %d\n", dev);
      idx = 0;
      while ((hipSetDevice(idx) != hipSuccess) && (idx < nDev)) { idx++; }
      if (idx == nDev) {
	printf("CUDAUTIL ERROR: Could not assign a CUDA device!\n"); exit(-333);
      } else {
	cudaCheckErr(hipGetDevice(&dev),__LINE__,__FILE__);
	info->device = dev;
	printf("CUDAUTIL WARNING: Re-assigning to CUDA device %d\n", dev);
      }
    } else {
      printf("Using CUDA Device %d\n", dev);
    }
  } else {
    idx = 0;
    while ((hipSetDevice(idx) != hipSuccess) && (idx < nDev)) { idx++; }
    if (idx == nDev) {
      printf("CUDAUTIL ERROR: Could not assign a CUDA device!\n"); exit(-334);
    } else {
      cudaCheckErr(hipGetDevice(&dev),__LINE__,__FILE__);
      info->device = dev;
      printf("Assigning to CUDA device %d\n", dev);
    }
  }

  // Process management and initialization
  if ((proc != -1) && (proc != 0)) {
    cudaCheckErr(hipStreamCreate((hipStream_t*)&proc),__LINE__,__FILE__);
    info->process = proc;
    printf("Assigning to CUDA Stream %d\n", proc);  
  } else {
    info->process = proc;
  }
  return;
}

/**
 * cudaInitMem function
 * 
 * \brief  This function intelligently allocates memory on the host or device
 *         as specified and returns pointer to the memory as well as the
 *         stride for the data, which is used for multi-dimensional data
 *         blocks.
 * \param  dims The dimensions of the memory to be allocated
 * \param  stride The stride (width in bytes) of the data. This only matters
 *         for multi-dimensional data
 * \param  loc The location of the memory to be allocated (host or device)
 * \param  ptr A pointer to where the memory pointer is to be stored
 * \return None
 *
 */
extern "C" void * cudaInitMem(int * dims, int typeSize, int * stride, const char * name, HTaskInfo * info, int mapHostFlag) {
  void * locPtr = NULL;
  size_t free, total = 0;
  int datSize = 1;
  int dev = info->device;
  hTaskLoc loc = info->location;

  // Set to proper device
  cudaCheckErr(hipSetDevice(dev),__LINE__,__FILE__);

  // Get full data size
  for (int idx = 0; idx < HNDIMS; idx++) { datSize *= dims[idx]; }
  datSize *= typeSize;

  if (!strcmp(name,"")) { // Data is not a symbol
    // CHECK 1: Memory requested available in global memory
    cudaCheckErr(hipMemGetInfo(&free, &total),__LINE__,__FILE__);
    if (free > datSize) { // There is adequate available memory
      // Allocate memory for an object
      hipExtent extent = make_hipExtent(dims[HLENGTH]*typeSize, dims[HWIDTH], dims[HDEPTH]);
      hipPitchedPtr pitchedPtr;
      cudaCheckErr(hipMalloc3D(&pitchedPtr, extent),__LINE__,__FILE__);
      cudaCheckErr(hipMemset3D(pitchedPtr, 0, extent),__LINE__,__FILE__);
      locPtr = pitchedPtr.ptr;
      *stride = pitchedPtr.pitch;
      cudaCheckErr(hipMemGetInfo(&free, &total),__LINE__,__FILE__);
      printf("Allocated %dB of memory on device. %dB remaining of %dB total\n", datSize, free, total);
      if (*stride != (dims[HLENGTH]*typeSize)) {
	printf("CUDAUTIL WARNING: Stride is not the same size as dims[HLENGTH]\n");
      }
    } else {
      printf("CUDAUTIL ERROR: Attempting to allocate %dB of memory when only %dB are \
         available out of a total of %dB\n", datSize, free, total); exit(-336);
    }
  } else { // Data is a CUDA symbol
    cudaCheckErr(hipGetSymbolAddress(&locPtr, HIP_SYMBOL(name)),__LINE__,__FILE__);
    cudaCheckErr(hipGetSymbolSize((size_t*)(&dims[HLENGTH]), name),__LINE__,__FILE__);
  }
  hipDeviceSynchronize();
  cudaCheckErr(hipGetLastError(),__LINE__,__FILE__);
  return locPtr;
}

/**
 * cudaFreeMem function
 * 
 * \brief  This function frees allocated memory on the host or device
 *         as specified.
 * \param  ptr Pointer to the data to be freed
 * \param  dims The dimensions of the data to be freed
 * \param  loc The location of the data to be freed
 * \return None
 *
 */
extern "C" void cudaFreeMem( void * ptr, int * dims, int typeSize, const char * name, HTaskInfo * info, int mapHostFlag ) {
  hTaskLoc loc = info->location;
  int dev = info->device;

  if (mapHostFlag) {
    cudaCheckErr(hipHostFree(ptr),__LINE__,__FILE__);
  } else {
    cudaCheckErr(hipSetDevice(dev),__LINE__,__FILE__);
    if (!strcmp(name,"")) { cudaCheckErr(hipFree(ptr),__LINE__,__FILE__); }
  }
  hipDeviceSynchronize();
  cudaCheckErr(hipGetLastError(),__LINE__,__FILE__);
  return;
}

/**
 * cudaClearMem function
 * 
 * \brief  
 * \param  dims The dimensions of the memory to be allocated
 * \param  stride The stride (width in bytes) of the data. This only matters
 *         for multi-dimensional data
 * \param  loc The location of the memory to be allocated (host or device)
 * \param  ptr A pointer to where the memory pointer is to be stored
 * \return None
 *
 */
extern "C" void cudaClearMem(int * dims, int typeSize, int stride, void * ptr, HTaskInfo * info) {
  int datSize = 0;
  int dev = info->device;
  hTaskLoc loc = info->location;

  // Set to proper device
  cudaCheckErr(hipSetDevice(dev),__LINE__,__FILE__);

  // Get full data size
  for (int idx = 0; idx < HNDIMS; idx++) { datSize += dims[idx]; }
  datSize *= typeSize;

  // Clear memory on the host for device use
  hipExtent extent = make_hipExtent(dims[HLENGTH]*typeSize, dims[HWIDTH], dims[HDEPTH]);
  hipPitchedPtr pitchedPtr;
  pitchedPtr.pitch = stride;
  pitchedPtr.ptr = ptr;
  pitchedPtr.xsize = dims[HLENGTH];
  pitchedPtr.ysize = dims[HWIDTH];
  cudaCheckErr(hipMemset3D(pitchedPtr, 0, extent),__LINE__,__FILE__);
  hipDeviceSynchronize();
  cudaCheckErr(hipGetLastError(),__LINE__,__FILE__);
  return;
}

/**
 * cudaMoveData function
 * 
 * \brief  This function intelligently copies data from the source location
 *         with the specified stride to the destination location with the 
 *         specified stride. The dimensions parameter indicates the
 *         dimensions of the data and must be the same for both source and
 *         destination.
 * \param  dest Pointer to the destination memory location
 * \param  destStride Stride (width in bytes) of the destination
 *         memory location
 * \param  destLoc The location of the destination memory
 * \param  src Pointer to the source memory location
 * \param  srcStride Stride (width in bytes) of the source memory
 *         location
 * \param  srcLoc The location of the destination memory
 * \param  dims The dimensions of the data to be copied
 * \return None
 *
 */
extern "C" void cudaMoveData(	void * dst, int dstStride, HTaskInfo * dstInfo,
				void * src, int srcStride, HTaskInfo * srcInfo,
				int * dims, int typeSize, const char * name) {
  hTaskLoc srcLoc = srcInfo->location;
  hTaskLoc dstLoc = dstInfo->location;
  int srcDev = srcInfo->device;
  int dstDev = dstInfo->device;
  int dev = -1;
  int proc = -1;
  hipMemcpy3DParms cpyParms = {0};
  hipStream_t stream;
  hipMemcpyKind kind;

  // Set to proper device
  if (srcLoc == LOC_CUDA) { dev = srcDev; }
  if (dstLoc == LOC_CUDA) { dev = dstDev; }
  if (dev != -1) { cudaCheckErr(hipSetDevice(dev),__LINE__,__FILE__); }

  if (!strcmp(name,"")) { // conduit is not for a symbol
    // Determine stream value
    if ((srcLoc == LOC_CUDA) && (dstLoc == LOC_CPU)) {
      proc = srcInfo->process;
      kind = hipMemcpyDeviceToHost;
    } else if ((dstLoc == LOC_CUDA) && (srcLoc == LOC_CPU)) {
      proc = dstInfo->process;
      kind = hipMemcpyHostToDevice;
    } else if ((dstLoc == LOC_CUDA) && (srcLoc == LOC_CUDA)) {
      proc = srcInfo->process;
      kind = hipMemcpyDeviceToDevice;
    } else if ((dstLoc == LOC_CPU) && (srcLoc == LOC_CPU)) {
      proc = srcInfo->process;
      kind = hipMemcpyHostToHost;
    }
    // Both src and dst are CPUs
    int stride = (srcStride > dstStride) ? srcStride : dstStride;
    cpyParms.extent = make_hipExtent(dims[HLENGTH]*typeSize, dims[HWIDTH], dims[HDEPTH]);
    cpyParms.srcPtr.pitch = stride;
    cpyParms.srcPtr.ptr = src;
    cpyParms.srcPtr.xsize = dims[HLENGTH]*typeSize;
    cpyParms.srcPtr.ysize = dims[HWIDTH];
    cpyParms.dstPtr.pitch = stride;
    cpyParms.dstPtr.ptr = dst;
    cpyParms.dstPtr.xsize = dims[HLENGTH]*typeSize;
    cpyParms.dstPtr.ysize = dims[HWIDTH];
    cpyParms.kind = kind;
    cudaCheckErr(hipMemcpy3DAsync(&cpyParms, (hipStream_t)(proc)),__LINE__,__FILE__);
    hipStreamSynchronize((hipStream_t)(proc));
  } else { // this is a symbol
    size_t datSize = dims[HLENGTH]*dims[HWIDTH]*dims[HDEPTH]*typeSize;
    if (dstLoc == LOC_CUDA) {
      hipMemcpyToSymbol(HIP_SYMBOL(name), src, datSize, 0, hipMemcpyHostToDevice);
    } else if (srcLoc == LOC_CUDA) {
      hipMemcpyFromSymbol(dst, HIP_SYMBOL(name), datSize, 0, hipMemcpyDeviceToHost);
    } else {
      printf("CUDAUTIL ERROR: Invalid device location for symbol\n"); exit(-340);
    }
  }
  hipDeviceSynchronize();
  cudaCheckErr(hipGetLastError(),__LINE__,__FILE__);
  return;
}

/**
 * hipLaunchKernel function
 * 
 * \brief  This function uses the specified parameters to execute the
 *         coprocessor function specified by kernel. Any parameters to
 *         the kernel should be included in params.
 * 
 * \param  kernel A string naming the kernel to execute
 * \param  dims The dimensions of the data to execute the kernel on
 * \param  nParams The number of parameters to be passed to the kernel
 * \param  params Array of parameters to be passed to the kernel
 * \param  gDim The grid dimensions of the kernel
 * \param  bDim The block dimensions of the kernel
 * \param  locMem The amount of local or shared memory to allocate for kernel
 *         execution
 * \param  stream The stream index to associate this kernel's execution and
 *         data with
 * \return None
 *
 */
/*extern "C" void hipLaunchKernel(reinterpret_cast<const void*>(	const char * krn), int * dims, int nParams, int * paramSizes, void ** params,
					int * gDim, int * bDim, int locMem, HTaskInfo * info) {
  hTaskLoc loc = info->location;
  int dev = info->device;
  int proc = info->process;
  size_t offset = 0;
  dim3 gridDims = dim3(gDim[HLENGTH],gDim[HWIDTH],gDim[HDEPTH]);
  dim3 blockDims = dim3(bDim[HLENGTH],bDim[HWIDTH],bDim[HDEPTH]);


  // Set the device and configure the kernel call to a stream
  cudaCheckErr(hipSetDevice(dev),__LINE__,__FILE__);
  cudaCheckErr(hipConfigureCall(gridDims, blockDims, locMem, (hipStream_t)proc),__LINE__,__FILE__);
	
  // For each kernel parameter passed in, push the kernel argument onto the stack
  //   turn into internal function
  float *s;
  hipMalloc(&s,sizeof(float)*2);
  params[0]=&s;
  for (int i = 0; i < nParams; i++) {
    cudaCheckErr(hipSetupArgument(params[i], paramSizes[i], offset),__LINE__,__FILE__);
    offset = offset + paramSizes[i];
  }
  
  if (!strcmp(krn,"scale")) {
    printf("Launching %s on dev:%d\n", krn, dev);
    cudaCheckErr(hipLaunchByPtr(scale),__LINE__,__FILE__);
  }

  cudaCheckErr(hipGetLastError(),__LINE__,__FILE__);

  if ((proc != 0) && (proc != -1)) { hipStreamSynchronize((hipStream_t)proc); }
  //hipDeviceSynchronize();
  cudaCheckErr(hipDeviceSynchronize(),__LINE__,__FILE__);
  printf("proc= %d  run to here\n", proc);
  cudaCheckErr(hipGetLastError(),__LINE__,__FILE__);
  return;
}
*/

